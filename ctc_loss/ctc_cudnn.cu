
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <iomanip>
#include "cudnn.h"

#define CUDNN_CALL(func)                                                       \
  {                                                                            \
    auto e = (func);                                                           \
    if (e != CUDNN_STATUS_SUCCESS) {                                           \
        std::cerr << "cuDNN error in " << __FILE__ << ":" << __LINE__;         \
        std::cerr << " : " << cudnnGetErrorString(e) << std::endl;             \
        exit(1);                                                               \
    }                                                                          \
  }

#define CUDA_CALL(func)                                                        \
  {                                                                            \
    auto e = (func);                                                           \
    if ((func) != hipSuccess) {                                               \
        std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__;          \
        std::cerr << " : " << hipGetErrorString(e) << std::endl;              \
        exit(1);                                                               \
    }                                                                          \
  }

#ifndef Dtype
#define Dtype float
#endif

int main() {
    const int kNumTimestamps = 4;
    const int kNumLabels = 3 + 1;
    const int kBatchSize = 5;

    cudnnHandle_t handle;
    CUDNN_CALL(cudnnCreate(&handle));

    auto cudnn_dtype = CUDNN_DATA_FLOAT;
    if (std::is_same_v<Dtype, double>) cudnn_dtype = CUDNN_DATA_DOUBLE;

    cudnnTensorDescriptor_t probs;
    Dtype* pProbs;
    {
      CUDNN_CALL(cudnnCreateTensorDescriptor(&probs));
      const int dims[] {kNumTimestamps, kBatchSize, kNumLabels};
      const int strides[] {kBatchSize * kNumLabels, kNumLabels, 1};
      CUDNN_CALL(cudnnSetTensorNdDescriptor(probs, cudnn_dtype, 3, dims,
                                            strides));
      int total_size = kNumLabels * kNumTimestamps * kBatchSize;
      CUDA_CALL(hipMallocManaged(&pProbs, sizeof(Dtype) * total_size));
      for(int i = 0; i < kNumTimestamps * kBatchSize; i++) {
        pProbs[i * kNumLabels + 0] = 1.f;
        pProbs[i * kNumLabels + 1] = 2.f;
        pProbs[i * kNumLabels + 2] = 3.f;
        pProbs[i * kNumLabels + 3] = 4.f;
        // pProbs[i * kNumLabels + 0] = 1.f;
        // pProbs[i * kNumLabels + 1] = 1.f;
        // pProbs[i * kNumLabels + 2] = 1.f;
        // pProbs[i * kNumLabels + 3] = 1.f;
      }
    }

    cudnnTensorDescriptor_t grads;
    Dtype* pGrads;
    {
      CUDNN_CALL(cudnnCreateTensorDescriptor(&grads));
      const int dims[] {kNumTimestamps, kBatchSize, kNumLabels};
      const int strides[] {kBatchSize * kNumLabels, kNumLabels, 1};
      CUDNN_CALL(cudnnSetTensorNdDescriptor(grads, cudnn_dtype, 3, dims,
                                            strides));
      int total_size = kNumLabels * kNumTimestamps * kBatchSize;
      CUDA_CALL(hipMalloc(&pGrads, sizeof(Dtype) * total_size));
    }

    cudnnCTCLossDescriptor_t ctcLossDesc;
    CUDNN_CALL(cudnnCreateCTCLossDescriptor(&ctcLossDesc));
    CUDNN_CALL(cudnnSetCTCLossDescriptorEx(ctcLossDesc, cudnn_dtype,
                                           CUDNN_LOSS_NORMALIZATION_SOFTMAX,
                                           // CUDNN_LOSS_NORMALIZATION_NONE,
                                           CUDNN_NOT_PROPAGATE_NAN));

    size_t workspace_size;

    int labels[] {1, 2, 3,
                  1, 2, 3,
                  1, 2, 3,
                  1, 2, 3,
                  1, 2, 3};
    int labelLengths[] {3, 3, 3, 3, 3};
    int inputLengths[] {4, 4, 4, 4, 4};

    CUDNN_CALL(cudnnGetCTCLossWorkspaceSize(
                   handle, probs, grads, labels, labelLengths, inputLengths,
                   CUDNN_CTC_LOSS_ALGO_DETERMINISTIC, ctcLossDesc,
                   &workspace_size));
    printf("Workspace in bytes: %ld\n", workspace_size);

    void *workspace;
    CUDA_CALL(hipMalloc(&workspace, workspace_size));

    Dtype *costs;
    CUDA_CALL(hipMalloc(&costs, sizeof(Dtype) * kBatchSize));

    CUDNN_CALL(cudnnCTCLoss(
                   /*handle*/handle,
                   /*probsDesc*/probs,
                   /*probs*/pProbs,
                   /*hostLabels*/labels,
                   /*hostLabelLengths*/labelLengths,
                   /*hostInputLengths*/inputLengths,
                   /*costs*/costs,
                   /*gradidentsDesc*/grads,
                   /*gradidents*/pGrads,
                   /*algo*/CUDNN_CTC_LOSS_ALGO_DETERMINISTIC,
                   /*ctcLossDesc*/ctcLossDesc,
                   /*workspace*/workspace,
                   /*workspaceSizeInBytes*/workspace_size));

    CUDA_CALL(hipDeviceSynchronize());

    {
      std::cout << "Loss: " << std::endl;
      Dtype* ptr = new Dtype[kBatchSize];
      CUDA_CALL(hipMemcpy(ptr, costs, sizeof(Dtype) * kBatchSize,
                hipMemcpyDeviceToHost));
      std::cout << std::fixed;
      std::cout << std::setprecision(8);
      for( size_t i = 0; i < kBatchSize; i++) {
        std::cout << ptr[i] << ", ";
      }
      std::cout << std::endl;
      delete[] ptr;
    }

    {
      std::cout << "Grads: " << std::endl;
      int total_size = kNumLabels * kNumTimestamps * kBatchSize;
      Dtype *ptr = new Dtype[total_size];
      CUDA_CALL(hipMemcpy(ptr, pGrads, sizeof(Dtype) * total_size,
                           hipMemcpyDeviceToHost));
      std::cout << std::fixed;
      std::cout << std::setprecision(8);
      for(int i = 0; i < kNumTimestamps; i++) {
        for(int j = 0; j < kBatchSize; j++) {
          for(int k = 0; k < kNumLabels; k++) {
            std::cout << ptr[i * kBatchSize * kNumLabels + j * kNumLabels + k]
                << ", ";
          }
          std::cout << std::endl;
        }
        std::cout << std::endl;
      }
      delete[] ptr;
    }

    hipFree(costs);
    hipFree(workspace);
    hipFree(pProbs);
    hipFree(pGrads);

    CUDNN_CALL(cudnnDestroyCTCLossDescriptor(ctcLossDesc));
    CUDNN_CALL(cudnnDestroyTensorDescriptor(probs));
    CUDNN_CALL(cudnnDestroyTensorDescriptor(grads));
    CUDNN_CALL(cudnnDestroy(handle));
}
